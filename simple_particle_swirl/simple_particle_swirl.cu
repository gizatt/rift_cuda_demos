#include "hip/hip_runtime.h"
/* #########################################################################
        simple_particle_swirl: Oculus Rift CUDA-powered demo 
   
   Interfaces with the RIFT SDK and CUDA to demo... well,
   both. Fancy particle graphics, what's not to like!

   read_from_due mockup to test reading from the arduino. 

   Rev history:
     Gregory Izatt  20130717  Init revision
   ######################################################################### */    

// Us!
#include "simple_particle_swirl.h"

// use protection guys
using namespace std;
using namespace OVR;

//GLUT:
// mouse controls (copied from CUDA SDK ocean example)
int mouseOldX, mouseOldY;
int mouseButtons = 0;
float rotateX = 20.0f, rotateY = 0.0f;
float translateX = 0.0f, translateY = 0.0f, translateZ = -2.0f;
int screenX, screenY;
//Frame counters
int totalFrames = 0;
int frame = 0; //Start with frame 0
unsigned long lastTicks_framerate;
unsigned long lastTicks_elapsed;
double currFrameRate = 0;
unsigned long perfFreq;

//VBO for particles
// vbo variables
static GLuint vbo[1];
hipGraphicsResource *resources[1];
// Device buffer variables
float4* d_velocities;

//Rift
Ptr<DeviceManager> pManager;
Ptr<HMDDevice> pHMD;
Ptr<SensorDevice> pSensor;
HMDInfo hmd;
SensorFusion SFusion;

/* #########################################################################
    
                            forward declarations
                            
   ######################################################################### */        
// opengl initialization
void initOpenGL(int w, int h, void*d);
//    GLUT display callback -- updates screen
void glut_display();
// GLUT idle callback -- launches a CUDA analysis cycle
void glut_idle();
//GLUT resize callback
void resize(int width, int height);
//Key handlers and mouse handlers; all callbacks for GLUT
void normal_key_handler(unsigned char key, int x, int y);
void special_key_handler(int key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

// Get our framerate
double get_framerate();
// Return curr time in ms since last call to this func (high res)
double get_elapsed();

// And the magnificent kernel!
__global__ void d_simple_particle_swirl( float4* pos, float4* vels, unsigned int N, float dt); 

/* #########################################################################
    
                                    MAIN
                                    
        -Parses cmdline args (none right now)
        -Initializes OpenGL and log file
        -%TODO some setup stuff!
        -Registers callback funcs with GLUT
        -Gives control to GLUT's main loop
        
   ######################################################################### */    
int main(int argc, char* argv[]) {    
    //Deal with cmd-line args
    if (argc!=1){
        printf("Usage: no arguments. That's it ¯\\_(ツ)_/¯\n");
        exit(1);
    }
    
    printf("Initializing... ");
    srand(time(0));
    //Set up timer
    LARGE_INTEGER li;
    if(!QueryPerformanceFrequency(&li))
        printf("QueryPerformanceFrequency failed!\n");
    perfFreq = (unsigned long)(li.QuadPart);
    //Rift init
    pManager = *DeviceManager::Create();
    printf("pManager: %p\n", pManager);
    if (pManager){
        pHMD = *pManager->EnumerateDevices<HMDDevice>().CreateDevice();
        printf("2/5\n");
        if (pHMD->GetDeviceInfo(&hmd))
        {
            char * MonitorName = hmd.DisplayDeviceName;
            printf("%s\n", MonitorName);
        }
        printf("3\n");
        if (pSensor)
            SFusion.AttachToSensor(pSensor);
        printf("4\n");
    }

    //Go get openGL set up / get the critical glob. variables set up
    initOpenGL(1024, 768, NULL);


    //Gotta register our callbacks
    glutIdleFunc( glut_idle );
    glutDisplayFunc( glut_display );
    glutKeyboardFunc ( normal_key_handler );
    glutSpecialFunc ( special_key_handler );
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutReshapeFunc(resize);

    //Main loop!
    printf("done!\n");
    glutMainLoop();

    return(1);
}


/* #########################################################################
    
                                initOpenGL
                                            
        -Sets up both a CUDA and OpenGL context
        -Initializes OpenGL for 3D rendering
        -Initializes the shared vertex buffer that we'll use, and 
            gets it registered with CUDA
        
   ######################################################################### */
void initOpenGL(int w, int h, void*d = NULL) {

    //Start off by resetting cudaDevice
    hipDeviceReset();
        
    // first, find a CUDA device and set it to graphic interop
    hipDeviceProp_t  prop;
    int dev;
    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    cudaGLSetGLDevice( dev );
        
    
    // a bug in the Windows GLUT implementation prevents us from
    // passing zero arguments to glutInit()
    int c=1;
    char* dummy = "";
    glutInit( &c, &dummy );
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH );
    glutInitWindowSize( w, h );
    glutCreateWindow( "display" );

    //Get glew set up, and make sure that worked
    GLenum err = glewInit();
    if (GLEW_OK != err)
    {
        /* Problem: glewInit failed, something is seriously wrong. */
        fprintf(stderr, "Error: %s\n", glewGetErrorString(err));
    }
        
    //Viewpoint setup
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    float ratio =  w * 1.0 / h;
    gluPerspective(45.0f, ratio, 0.1f, 100.0f);
    glMatrixMode(GL_MODELVIEW);
    
    //And adjust point size
    glPointSize(2);
    //Enable depth-sorting of points during rendering
    glDepthFunc(GL_LESS);
    glEnable(GL_DEPTH_TEST);
    glEnable (GL_BLEND);
    glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    //Clear viewport
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);  
    
    //And set up shared vertex buffer
    glGenBuffers( 1, vbo );
    glBindBuffer( GL_ARRAY_BUFFER, *vbo );
    float4* temppos = (float4*)malloc(BUFFER_SIZE);
    if (!temppos){ printf("Memory alloc error.\n"); exit(1);}
    for(int i = 0; i < NUM_PARTICLES; i++)
    {
        /* Initial position within 0.05 of <0, 0.5, 0> */
        temppos[i].x = ((float)rand())/RAND_MAX * 0.1 - 0.05;
        temppos[i].y = ((float)rand())/RAND_MAX * 0.1 + 0.45;
        temppos[i].z = ((float)rand())/RAND_MAX * 0.1 - 0.05;
        temppos[i].w = 1.0;
    }
    glBufferData( GL_ARRAY_BUFFER, BUFFER_SIZE, temppos, GL_DYNAMIC_DRAW );
    // (whenever I bind buffer index 0, that's just the way of unbinding
    //     openGL from any buffer...)
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    free(temppos);
    if (glGetError()){
        unsigned char * glErrorBuffer = (unsigned char *) gluErrorString(glGetError());
        printf("Opengl error: %s\n", glErrorBuffer);
    }
    // register, map and unmap to try to cycle it in...
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(resources, *vbo, cudaGraphicsMapFlagsWriteDiscard) );

    // allocate velocity buffer on device side
    CUDA_SAFE_CALL( hipMalloc( (void**)&d_velocities, BUFFER_SIZE ) );
    float4* tempvel = (float4*)malloc(BUFFER_SIZE);
    if (!tempvel){ printf("Memory alloc error.\n"); exit(1);}
    for(int i = 0; i < NUM_PARTICLES; i++)
    {
        /* Initial velocity near <1, 0, 0> */
        tempvel[i].x = ((float)rand())/RAND_MAX * 0.15 + 0.950;
        tempvel[i].y = ((float)rand())/RAND_MAX * 0.05 - 0.025;
        tempvel[i].z = ((float)rand())/RAND_MAX * 0.05 - 0.025;
        tempvel[i].w = 1.0;
    }
    CUDA_SAFE_CALL( hipMemcpy( d_velocities, tempvel, BUFFER_SIZE, hipMemcpyHostToDevice ) );
    free(tempvel);
    //store our screen sizing information
    screenX = glutGet(GLUT_WINDOW_WIDTH);
    screenY = glutGet(GLUT_WINDOW_HEIGHT);



    glFinish();
}

void resize(int width, int height){
    glViewport(0,0,width,height);
    screenX = glutGet(GLUT_WINDOW_WIDTH);
    screenY = glutGet(GLUT_WINDOW_HEIGHT);
}


/* #########################################################################
    
                                glut_display
                                            
        -Callback from GLUT: called whenever screen needs to be
            re-rendered
        -Feeds vertex buffer, along with specifications of what
            is in it, to OpenGL to render.
        
   ######################################################################### */    
void render_core(){

}
void glut_display(){

    //Clear out buffers before rendering the new scene
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // bring in vbo
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    //Left viewport:
    glViewport(0,0,screenX/2,screenY);
    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    // reset it to default
    glLoadIdentity();
    // And transform in camera position
    glTranslatef(translateX+0.1, translateY, translateZ);
    glRotatef(rotateX, 1.0, 0.0, 0.0);
    glRotatef(rotateY, 0.0, 1.0, 0.0);
    // render from the vbo
    glVertexPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, NUM_PARTICLES);
    glDisableClientState(GL_VERTEX_ARRAY);
    
    //Right viewport:
    glViewport(screenX/2,0,screenX/2,screenY);
    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    // reset it to default
    glLoadIdentity();
    // And transform in camera position
    glTranslatef(translateX-0.1, translateY, translateZ);
    glRotatef(rotateX, 1.0, 0.0, 0.0);
    glRotatef(rotateY, 0.0, 1.0, 0.0);
    // render from the vbo
    glVertexPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, NUM_PARTICLES);
    glDisableClientState(GL_VERTEX_ARRAY);

    // swap whole screen
    glutSwapBuffers();   
    // free vbo for CUDA
    glBindBuffer(GL_ARRAY_BUFFER, 0);


    double curr = get_framerate();
    if (currFrameRate != 0.0f)
        currFrameRate = (10.0f*currFrameRate + curr)/11.0f;
    else
        currFrameRate = curr;

    //output useful framerate and status info:
    printf ("framerate: %3.1f / %4.1f\n", curr, currFrameRate);
    
    //Unbind, restore cuda access to buffer, and continue:
    /*
    glDisableClientState( GL_COLOR_ARRAY );
    glDisableClientState( GL_VERTEX_ARRAY );
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    
    HANDLE_ERROR( hipGraphicsMapResources(1, resources, fftStream) );
    size_t size;
    HANDLE_ERROR( hipGraphicsResourceGetMappedPointer((void **)(&vertexPointer), &size, resources[0]) );
    
    if (writelog) fprintf(log_file, "%05.5f, Exit refresh_screen\n", get_elapsed());
    if (writelog) fflush(log_file);
    */
    totalFrames++;
}


/* #########################################################################
    
                                glut_idle
                                            
        -Callback from GLUT: called as the idle function, as rapidly
            as possible.
        - Grab VBO control for CUDA, run kernel to advance particles,
            and pass VBO control back when done.

   ######################################################################### */    
int framesRendered = 0;
void glut_idle(){
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    CUDA_SAFE_CALL( hipGraphicsMapResources(1, resources) );
    size_t size;
    CUDA_SAFE_CALL( hipGraphicsResourceGetMappedPointer((void **)(&dptr), &size, resources[0]) );
    float dt = (float) get_elapsed();

    // execute the kernel
    if (framesRendered > 0) 
        d_simple_particle_swirl<<< GRID_SIZE, BLOCK_SIZE >>>(dptr, d_velocities, NUM_PARTICLES, dt);

    // unmap buffer object
    CUDA_SAFE_CALL(hipGraphicsUnmapResources(1, resources, 0));
    framesRendered++;
    glutPostRedisplay();
}


/* #########################################################################
    
                              normal_key_handler
                              
        -GLUT callback for non-special (generic letters and such)
          keypresses.
        -If escape is pressed, cleans up and exits.
        
   ######################################################################### */    
void normal_key_handler(unsigned char key, int x, int y) {

    switch (key) {
        default:
            printf("Unhandled key %u\n", key);
            break;
    }
}


/* #########################################################################
    
                              special_key_handler
                              
        -GLUT callback for special (arrow keys, F keys, etc)
          keypresses.
        -Binds up/down to adjusting parameters
        
   ######################################################################### */    
void special_key_handler(int key, int x, int y){
    switch (key) {
        default:
            printf("Unhandled key %u\n", key);
            break;
    }
}


/* #########################################################################
    
                                    mouse
                              
        -GLUT callback for mouse button presses
        -Records mouse button presses when they happen, for use
            in the mouse movement callback, which does the bulk of
            the work in managing the camera
        
   ######################################################################### */    
void mouse(int button, int state, int x, int y){
    if (state == GLUT_DOWN) {
        mouseButtons |= 1<<button;
    } else if (state == GLUT_UP) {
        mouseButtons = 0;
    }

    mouseOldX = x;
    mouseOldY = y;
}


/* #########################################################################
    
                                    motion
                              
        -GLUT callback for mouse motion
        -When the mouse moves and various buttons are down, adjusts camera.
        
   ######################################################################### */    
void motion(int x, int y){
    float dx, dy;
    dx = (float)(x - mouseOldX);
    dy = (float)(y - mouseOldY);

    if (mouseButtons == 1) {
        rotateX += dy * 0.2f;
        rotateY += dx * 0.2f;
    } else if (mouseButtons == 2) {
        translateX += dx * 0.01f;
        translateY -= dy * 0.01f;        
    } else if (mouseButtons == 4) {
        translateZ += dy * 0.01f;
    }

    mouseOldX = x;
    mouseOldY = y;
}


/* #########################################################################
    
                                get_framerate
                                            
        -Takes totalFrames / ((curr time - start time)/CLOCKS_PER_SEC)
            INDEPENDENT OF GET_ELAPSED; USING THESE FUNCS FOR DIFFERENT
                TIMING PURPOSES
   ######################################################################### */     
double get_framerate ( ) {
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    unsigned long currTicks = (unsigned long)(li.QuadPart);
    unsigned long elapsed = currTicks - lastTicks_framerate;
    double ret;
    if (elapsed != 0){
        ret = (((double)totalFrames) / (((double)elapsed)/((double)perfFreq)));
    }else{
        ret =  -1;
    }
    totalFrames = 0;
    lastTicks_framerate = currTicks;
    return ret;
}


/* #########################################################################
    
                                get_elapsed
                                            
        -Returns number of milliseconds since last call to this func
            as a double
        
   ######################################################################### */ 
double get_elapsed(){
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    unsigned long currTicks = (unsigned long)(li.QuadPart);
    double elapsed = (((double)(currTicks - lastTicks_elapsed))/((double)perfFreq))*1000;
    lastTicks_elapsed = currTicks;
    return elapsed;
}




/* #########################################################################
    
                           d_simple_particle_swirl
                                    KERNEL!
        
   ######################################################################### */ 
__global__ void d_simple_particle_swirl(float4* pos, float4* vels, unsigned int N, float dt)
{
    // Indices into the VBO data.
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N) {
        /* Update vels to orbit origin */
        float dist2 = pos[i].x*pos[i].x + pos[i].y*pos[i].y + pos[i].z*pos[i].z;
        if (dist2 != 0){
            vels[i].x -= 0.01 * pos[i].x / dist2;
            vels[i].y -= 0.01 * pos[i].y / dist2;
            vels[i].z -= 0.01 * pos[i].z / dist2;
        }
        /* And update position based on velocity */
        pos[i].x += vels[i].x*dt/1000.0;
        pos[i].y += vels[i].y*dt/1000.0;
        pos[i].z += vels[i].z*dt/1000.0;
    }
}
