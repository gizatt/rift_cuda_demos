#include "hip/hip_runtime.h"
/* #########################################################################
        simple_particle_swirl: Oculus Rift CUDA-powered demo 
   
   Interfaces with the RIFT SDK and CUDA to demo... well,
   both. Fancy particle graphics, what's not to like!

   Organized fairly haphazardly at the moment (class-ification to come
   in the future, probably), but it's based on glut's callback system.
   Function prototypes are listed below; main() does primarily initialization
   of the various helper classes and rift sdk things that we need to get
   up and running, sets up glut, and goes. Other functions do what they
   advertise, either drawing, passing idle ticks to our helper objects,
   passing input to helper objects, etc.

   Rev history:
     Gregory Izatt  20130717  Init revision
   ######################################################################### */    

// Us!
#include "simple_particle_swirl.h"
// And a helper player class
#include "../common/player.h"
#include "../common/rift.h"
// handy image loading
#include "../include/SOIL.h"

// use protection guys
using namespace std;
using namespace OVR;
using namespace xen_rift;

//Global light direction
float3 light_direction;

//GLUT:
int screenX, screenY;
//Frame counters
int totalFrames = 0;
int frame = 0; //Start with frame 0
unsigned long lastTicks_framerate;
unsigned long lastTicks_elapsed;
double currFrameRate = 0;
unsigned long perfFreq;

//VBO for particles
// vbo variables
static GLuint vbo[1];
hipGraphicsResource *resources[1];
// Device buffer variables
float4* d_velocities;

// ground tex
GLuint ground_tex;

/*
Ptr<DeviceManager> pManager;
Ptr<HMDDevice> pHMD;
Ptr<SensorDevice> pSensor;
HMDInfo hmd;
SensorFusion SFusion;
*/

//Player manager
Player * player_manager;
//Rift
Rift * rift_manager;


/* #########################################################################
    
                            forward declarations
                            
   ######################################################################### */        
// opengl initialization
void initOpenGL(int w, int h, void*d);
//    GLUT display callback -- updates screen
void glut_display();
// Helper to draw the demo room itself
void draw_demo_room();
// and shared between eyes rendering core
void render_core();
// GLUT idle callback -- launches a CUDA analysis cycle
void glut_idle();
//GLUT resize callback
void resize(int width, int height);
//Key handlers and mouse handlers; all callbacks for GLUT
void normal_key_handler(unsigned char key, int x, int y);
void normal_key_up_handler(unsigned char key, int x, int y);
void special_key_handler(int key, int x, int y);
void special_key_up_handler(int key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

// Get our framerate
double get_framerate();
// Return curr time in ms since last call to this func (high res)
double get_elapsed();

// And the magnificent kernel!
__global__ void d_simple_particle_swirl( float4* pos, float4* vels, unsigned int N, float dt,
        float3 player_pos, float3 light_dir); 

/* #########################################################################
    
                                    MAIN
                                    
        -Parses cmdline args (none right now)
        -Initializes OpenGL and log file
        -%TODO some setup stuff!
        -Registers callback funcs with GLUT
        -Gives control to GLUT's main loop
        
   ######################################################################### */    
int main(int argc, char* argv[]) {    
    //Deal with cmd-line args
    if (argc!=1){
        printf("Usage: no arguments. That's it ¯\\_(ツ)_/¯\n");
        exit(1);
    }
    
    printf("Initializing... ");
    srand(time(0));
    //Set up timer
    LARGE_INTEGER li;
    if(!QueryPerformanceFrequency(&li))
        printf("QueryPerformanceFrequency failed!\n");
    perfFreq = (unsigned long)(li.QuadPart);

    /*
    pManager = *DeviceManager::Create();
    printf("pManager: %p\n", pManager);
    if (pManager){
        pHMD = *pManager->EnumerateDevices<HMDDevice>().CreateDevice();
        printf("2/5\n");
        if (pHMD->GetDeviceInfo(&hmd))
        {
            char * MonitorName = hmd.DisplayDeviceName;
            printf("%s\n", MonitorName);
        }
        printf("3\n");
        if (pSensor)
            SFusion.AttachToSensor(pSensor);
        printf("4\n");
    }
    */

    //Go get openGL set up / get the critical glob. variables set up
    initOpenGL(1280, 720, NULL);

    // get helpers set up now that opengl is up
    float3 zeropos = float3();
    float2 zerorot = float2();
    player_manager = new Player(zeropos, zerorot, 1.6);
    //Rift
    rift_manager = new Rift(1280, 720, true);

    //Gotta register our callbacks
    glutIdleFunc( glut_idle );
    glutDisplayFunc( glut_display );
    glutKeyboardFunc ( normal_key_handler );
    glutKeyboardUpFunc ( normal_key_up_handler );
    glutSpecialFunc ( special_key_handler );
    glutSpecialUpFunc ( special_key_up_handler );
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutReshapeFunc(resize);

    //Main loop!
    printf("done!\n");
    glutMainLoop();

    return(1);
}


/* #########################################################################
    
                                initOpenGL
                                            
        -Sets up both a CUDA and OpenGL context
        -Initializes OpenGL for 3D rendering
        -Initializes the shared vertex buffer that we'll use, and 
            gets it registered with CUDA
        
   ######################################################################### */
void initOpenGL(int w, int h, void*d = NULL) {

    //Start off by resetting cudaDevice
    hipDeviceReset();
        
    // first, find a CUDA device and set it to graphic interop
    hipDeviceProp_t  prop;
    int dev;
    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    cudaGLSetGLDevice( dev );
        
    
    // a bug in the Windows GLUT implementation prevents us from
    // passing zero arguments to glutInit()
    int c=1;
    char* dummy = "";
    glutInit( &c, &dummy );
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH );
    glutInitWindowSize( w, h );
    glutCreateWindow( "display" );

    //Get glew set up, and make sure that worked
    GLenum err = glewInit();
    if (GLEW_OK != err)
    {
        /* Problem: glewInit failed, something is seriously wrong. */
        fprintf(stderr, "Error: %s\n", glewGetErrorString(err));
    }
    if (glewIsSupported("GL_VERSION_3_3"))
        ;
    else {
        printf("OpenGL 3.3 not supported\n");
        exit(1);
    }

    // Load in ground texture
    glEnable (GL_TEXTURE_2D);
    ground_tex = SOIL_load_OGL_texture
    (
        "../resources/groundgrid.bmp",
        SOIL_LOAD_AUTO,
        SOIL_CREATE_NEW_ID,
        SOIL_FLAG_MIPMAPS
    );
    if( 0 == ground_tex )
    {
        printf( "SOIL loading error: '%s'\n", SOIL_last_result() );
    }
    glBindTexture(GL_TEXTURE_2D, ground_tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri (GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri (GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri (GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_MODULATE);
    glBindTexture(GL_TEXTURE_2D, 0);

    // %TODO: this should talk to rift manager
    //Viewpoint setup
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    float ratio =  w * 1.0 / h;
    gluPerspective(45.0f, ratio, 0.1f, 100.0f);
    glMatrixMode(GL_MODELVIEW);
    
    //And adjust point size
    glPointSize(2);
    //Enable depth-sorting of points during rendering
    glDepthFunc(GL_LESS);
    glEnable(GL_DEPTH_TEST);
    glEnable (GL_BLEND);
    glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    //Define lighting
    light_direction = make_float3(0.5, -1.0, 0.0);
    light_direction = normalize(light_direction);
    GLfloat tmp[3] = {light_direction.x, light_direction.y, light_direction.z};
    GLfloat amb[]= { 0.1f, 0.1f, 0.1f, 1.0f };
    GLfloat diff[]= { 0.4f, 0.4f, 0.4f, 1.0f };
    GLfloat spec[]= { 0.8f, 0.8f, 0.8f, 1.0f };
    GLfloat lightpos[]= { 10.0f, 10.0f, 10.0f, 1.0f };
    glLightfv(GL_LIGHT0, GL_AMBIENT, amb);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, diff);
    glLightfv(GL_LIGHT0, GL_SPECULAR, spec);
    glLightfv(GL_LIGHT0, GL_POSITION, lightpos);
    // Turn on lighting.  You can turn it off with a similar call to
    // glDisable().
    glEnable(GL_LIGHTING);    
    glEnable(GL_LIGHT0);

    //Clear viewport
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);  
    
    //And set up shared vertex buffer
    glGenBuffers( 1, vbo );
    glBindBuffer( GL_ARRAY_BUFFER, *vbo );
    float4* temppos = (float4*)malloc(BUFFER_SIZE);
    if (!temppos){ printf("Memory alloc error.\n"); exit(1);}
    for(int i = 0; i < NUM_PARTICLES; i++)
    {
        /* Initial position in 30-radius ring at y=30 */
        float radius = ((float)rand())/RAND_MAX*27.0+3.0;
        float theta = ((float)rand())/RAND_MAX*2.0*M_PI/8.0;
        temppos[i].x = radius*cosf(theta);
        temppos[i].y = ((float)rand())/RAND_MAX * 1.0 + 29.5;
        temppos[i].z = radius*sinf(theta);
        unsigned char * tmp = (unsigned char *)&(temppos[i].w);
        tmp[0] = (unsigned char) 200;
        tmp[1] = (unsigned char) 200;
        tmp[2] = (unsigned char) 200;
        tmp[3] = 255;
    }
    glBufferData( GL_ARRAY_BUFFER, BUFFER_SIZE, temppos, GL_DYNAMIC_DRAW );
    // (whenever I bind buffer index 0, that's just the way of unbinding
    //     openGL from any buffer...)
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    if (glGetError()){
        unsigned char * glErrorBuffer = (unsigned char *) gluErrorString(glGetError());
        printf("Opengl error: %s\n", glErrorBuffer);
    }
    // register, map and unmap to try to cycle it in...
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(resources, *vbo, cudaGraphicsMapFlagsWriteDiscard) );

    // allocate velocity buffer on device side
    CUDA_SAFE_CALL( hipMalloc( (void**)&d_velocities, BUFFER_SIZE ) );
    float4* tempvel = (float4*)malloc(BUFFER_SIZE);
    if (!tempvel){ printf("Memory alloc error.\n"); exit(1);}
    for(int i = 0; i < NUM_PARTICLES; i++)
    {
        /* Initial velocity around origin at 0, 30, 0 */
        tempvel[i].x = ((float)rand())/RAND_MAX * 2.0 - 1.0 + temppos[i].z;
        tempvel[i].y = ((float)rand())/RAND_MAX * 1.0 - 0.5;
        tempvel[i].z = ((float)rand())/RAND_MAX * 2.0 - 1.0 - temppos[i].x;
        tempvel[i].w = 1.0;
    }
    CUDA_SAFE_CALL( hipMemcpy( d_velocities, tempvel, BUFFER_SIZE, hipMemcpyHostToDevice ) );
    free(tempvel);
    free(temppos);
    //store our screen sizing information
    screenX = glutGet(GLUT_WINDOW_WIDTH);
    screenY = glutGet(GLUT_WINDOW_HEIGHT);

    glEnable( GL_NORMALIZE );
    glFinish();
}

void resize(int width, int height){
    glViewport(0,0,width,height);
    screenX = glutGet(GLUT_WINDOW_WIDTH);
    screenY = glutGet(GLUT_WINDOW_HEIGHT);
}


/* #########################################################################
    
                                glut_display
                                            
        -Callback from GLUT: called whenever screen needs to be
            re-rendered
        -Feeds vertex buffer, along with specifications of what
            is in it, to OpenGL to render.
        
   ######################################################################### */    
void glut_display(){

    //Clear out buffers before rendering the new scene
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // bring in vbo
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);
    // and get player location
    float3 curr_translation = player_manager->get_position();
    float2 curr_rotation = player_manager->get_rotation();
    Vector3f curr_t_vec(curr_translation.x, curr_translation.y, curr_translation.z);
    Vector3f curr_r_vec(0.0f, curr_rotation.y*M_PI/180.0, 0.0f);

    // Go do Rift rendering!
    rift_manager->render(curr_t_vec, curr_r_vec, render_core);

    /*
    //Left viewport:
    glViewport(0,0,screenX/2,screenY);
    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    // reset it to default
    glLoadIdentity();
    // And transform in camera position
    glRotatef(curr_rotation.x, 1.0, 0.0, 0.0);
    glRotatef(curr_rotation.y, 0.0, 1.0, 0.0);   
    glTranslatef(curr_translation.x-0.1, curr_translation.y, curr_translation.z);
    // render from the vbo
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    // Each 8-byte vertex in that buffer includes coodinate information
    //  and color information:
    //  byte index [ 0  1  2  3  4  5  6  7  8  9  10  11  12  13  14  15  ]
    //  info       [ <x, float>  <y, float>  <z, float >   <r   g   b   a> ]
    // These cmds instruct opengl to expect that:
    glColorPointer(4,GL_UNSIGNED_BYTE,16,(void*)12);
    glVertexPointer(3,GL_FLOAT,16,(void*)0);
    glDrawArrays(GL_POINTS,0, NUM_PARTICLES);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    draw_demo_room();
    
    //Right viewport:
    glViewport(screenX/2,0,screenX/2,screenY);
    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    // reset it to default
    glLoadIdentity();
    // And transform in camera position
    glRotatef(curr_rotation.x, 1.0, 0.0, 0.0);
    glRotatef(curr_rotation.y, 0.0, 1.0, 0.0);
    glTranslatef(curr_translation.x+0.1, curr_translation.y, curr_translation.z);
    // render from the vbo
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    // Each 8-byte vertex in that buffer includes coodinate information
    //  and color information:
    //  byte index [ 0  1  2  3  4  5  6  7  8  9  10  11  12  13  14  15  ]
    //  info       [ <x, float>  <y, float>  <z, float >   <r   g   b   a> ]
    // These cmds instruct opengl to expect that:
    glColorPointer(4,GL_UNSIGNED_BYTE,16,(void*)12);
    glVertexPointer(3,GL_FLOAT,16,(void*)0);
    glDrawArrays(GL_POINTS,0, NUM_PARTICLES);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    draw_demo_room();


    // swap whole screen
    glutSwapBuffers();   
    */
    // free vbo for CUDA
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    double curr = get_framerate();
    if (currFrameRate != 0.0f)
        currFrameRate = (10.0f*currFrameRate + curr)/11.0f;
    else
        currFrameRate = curr;

    //output useful framerate and status info:
    //printf ("framerate: %3.1f / %4.1f\n", curr, currFrameRate);
    // frame was rendered, give the player handler a tick
    player_manager->on_frame_render();

    totalFrames++;
}

/* #########################################################################
    
                                draw_demo_room
                                            
        -Helper to draw demo room: basic walls, floor, environ, etc.
            If this becomes fancy enough / dynamic I may go throw it in
            its own file.

   ######################################################################### */
void draw_demo_room(){
    const float groundColor[]     = {0.7f, 0.7f, 0.7f, 1.0f};
    const float groundSpecular[]  = {0.1f, 0.1f, 0.1f, 1.0f};
    const float groundShininess[] = {0.10f};
    glEnable(GL_LIGHTING);
    glEnable(GL_TEXTURE_2D);
    //glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, ground_tex);

    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE, groundColor);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, groundSpecular);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SHININESS, groundShininess);

    glBegin(GL_QUADS);
    /* Floor */
    //glColor3f(1.0, 1.0, 1.0);
    glTexCoord2f (-10.0, -10.0);
    glVertex3f(-100.0,-0.1,-100.0);
    glTexCoord2f (10.0, -10.0);
    glVertex3f(100.0,-0.1,-100.0);
    glTexCoord2f (10.0, 10.0);
    glVertex3f(100.0,-0.1,100.0);
    glTexCoord2f (-10.0, 10.0);
    glVertex3f(-100.0,-0.1,100.0);
    glEnd();
    glDisable(GL_TEXTURE_2D);
}

/* #########################################################################
    
                                render_core
        Render functionality shared between eyes.

   ######################################################################### */
void render_core(){

    glEnable(GL_LIGHTING);
    glEnable(GL_DEPTH_TEST);

    const float partColor[]     = {0.9f, 0.1f, 0.1f, 1.0f};
    const float partSpecular[]  = {0.0f, 0.0f, 0.0f, 1.0f};
    const float partShininess[] = {0.0f};

    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE, partColor);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, partSpecular);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SHININESS, partShininess);
    // render from the vbo
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    // Each 8-byte vertex in that buffer includes coodinate information
    //  and color information:
    //  byte index [ 0  1  2  3  4  5  6  7  8  9  10  11  12  13  14  15  ]
    //  info       [ <x, float>  <y, float>  <z, float >   <r   g   b   a> ]
    // These cmds instruct opengl to expect that:
    glColorPointer(4,GL_UNSIGNED_BYTE,16,(void*)12);
    glVertexPointer(3,GL_FLOAT,16,(void*)0);
    glDrawArrays(GL_POINTS,0, NUM_PARTICLES);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    draw_demo_room();

    // draw in front-guide
    player_manager->draw_HUD();

    glDisable(GL_LIGHTING);
    glDisable(GL_DEPTH_TEST);
}

/* #########################################################################
    
                                glut_idle
                                            
        -Callback from GLUT: called as the idle function, as rapidly
            as possible.
        - Grab VBO control for CUDA, run kernel to advance particles,
            and pass VBO control back when done.

   ######################################################################### */    
int framesRendered = 0;
void glut_idle(){
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    CUDA_SAFE_CALL( hipGraphicsMapResources(1, resources) );
    size_t size;
    CUDA_SAFE_CALL( hipGraphicsResourceGetMappedPointer((void **)(&dptr), &size, resources[0]) );
    float dt = (float) get_elapsed();

    // execute the kernel
    if (framesRendered > 0) 
        d_simple_particle_swirl<<< GRID_SIZE, BLOCK_SIZE >>>(dptr, d_velocities, NUM_PARTICLES, dt,
            player_manager->get_position(), light_direction);

    // unmap buffer object
    CUDA_SAFE_CALL(hipGraphicsUnmapResources(1, resources, 0));

    // and let rift handler update
    rift_manager->onIdle();

    framesRendered++;
    glutPostRedisplay();
}


/* #########################################################################
    
                              normal_key_handler
                              
        -GLUT callback for non-special (generic letters and such)
          keypresses and releases.
        
   ######################################################################### */    
void normal_key_handler(unsigned char key, int x, int y) {
    player_manager->normal_key_handler(key, x, y);
    rift_manager->normal_key_handler(key, x, y);
    switch (key) {
        default:
            break;
    }
}
void normal_key_up_handler(unsigned char key, int x, int y) {
    player_manager->normal_key_up_handler(key, x, y);
    rift_manager->normal_key_up_handler(key, x, y);
    switch (key) {
        default:
            break;
    }
}

/* #########################################################################
    
                              special_key_handler
                              
        -GLUT callback for special (arrow keys, F keys, etc)
          keypresses and releases.
        -Binds up/down to adjusting parameters
        
   ######################################################################### */    
void special_key_handler(int key, int x, int y){
    player_manager->special_key_handler(key, x, y);
    rift_manager->special_key_handler(key, x, y);
    switch (key) {
        default:
            break;
    }
}
void special_key_up_handler(int key, int x, int y){
    player_manager->special_key_up_handler(key, x, y);
    rift_manager->special_key_up_handler(key, x, y);
    switch (key) {
        default:
            break;
    }
}

/* #########################################################################
    
                                    mouse
                              
        -GLUT callback for mouse button presses
        -Records mouse button presses when they happen, for use
            in the mouse movement callback, which does the bulk of
            the work in managing the camera
        
   ######################################################################### */    
void mouse(int button, int state, int x, int y){
    player_manager->mouse(button, state, x, y);
    rift_manager->mouse(button, state, x, y);
}


/* #########################################################################
    
                                    motion
                              
        -GLUT callback for mouse motion
        -When the mouse moves and various buttons are down, adjusts camera.
        
   ######################################################################### */    
void motion(int x, int y){
    player_manager->motion(x, y);
    rift_manager->motion(x, y);
}


/* #########################################################################
    
                                get_framerate
                                            
        -Takes totalFrames / ((curr time - start time)/CLOCKS_PER_SEC)
            INDEPENDENT OF GET_ELAPSED; USING THESE FUNCS FOR DIFFERENT
                TIMING PURPOSES
   ######################################################################### */     
double get_framerate ( ) {
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    unsigned long currTicks = (unsigned long)(li.QuadPart);
    unsigned long elapsed = currTicks - lastTicks_framerate;
    double ret;
    if (elapsed != 0){
        ret = (((double)totalFrames) / (((double)elapsed)/((double)perfFreq)));
    }else{
        ret =  -1;
    }
    totalFrames = 0;
    lastTicks_framerate = currTicks;
    return ret;
}


/* #########################################################################
    
                                get_elapsed
                                            
        -Returns number of milliseconds since last call to this func
            as a double
        
   ######################################################################### */ 
double get_elapsed(){
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    unsigned long currTicks = (unsigned long)(li.QuadPart);
    double elapsed = (((double)(currTicks - lastTicks_elapsed))/((double)perfFreq))*1000;
    lastTicks_elapsed = currTicks;
    return elapsed;
}




/* #########################################################################
    
                           d_simple_particle_swirl
                                    KERNEL!
        
   ######################################################################### */ 
__global__ void d_simple_particle_swirl(float4* pos, float4* vels, unsigned int N, float dt,
        float3 player_pos, float3 light_dir)
{
    // Indices into the VBO data.
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N) {
        /* Update vels to orbit <0, 3, 0> */
        float dist2 = pos[i].x*pos[i].x + pos[i].y*pos[i].y + pos[i].z*pos[i].z;
        if (dist2 != 0){
            vels[i].x -= 10.0 * pos[i].x / dist2;
            vels[i].y -= 10.0 * (pos[i].y - 30.0) / dist2;
            vels[i].z -= 10.0 * pos[i].z / dist2;
        }
        /* And update position based on velocity */
        pos[i].x += vels[i].x*dt/1000.0;
        pos[i].y += vels[i].y*dt/1000.0;
        pos[i].z += vels[i].z*dt/1000.0;
        /* Assign color as a rough diffuse lighting model */
        float3 to_our_pos = normalize(make_float3(pos[i].x - player_pos.x,
                                                  pos[i].y - player_pos.y,
                                                  pos[i].z - player_pos.z));
        float value = dot(to_our_pos, light_dir)*100.0+150.0;
        unsigned char * tmp = (unsigned char *)&(pos[i]);
        tmp[12] = (unsigned char)(value);
        tmp[13] = (unsigned char)(50); 
        tmp[14] = (unsigned char)(10); 
        tmp[15] = (unsigned char) 255;
    }
}
