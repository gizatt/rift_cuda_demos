#include "hip/hip_runtime.h"
/* #########################################################################
        simple_particle_swirl: Oculus Rift CUDA-powered demo kernels

   Rev history:
     Gregory Izatt  20130717  Init revision
     Gregory Izatt  20130814  Separating device / host code
   ######################################################################### */    

// Us!
#include "simple_particle_swirl_cu.h"

// use protection guys
using namespace std;
using namespace xen_rift;

//Frame counters
static int totalFrames = 0;
static int frame = 0; //Start with frame 0
static unsigned long lastTicks_framerate;
static unsigned long lastTicks_elapsed;
static double currFrameRate = 0;
static unsigned long perfFreq;

//VBO holder
hipGraphicsResource *resources[1];
// Device buffer variables
float4* d_velocities;

/* #########################################################################
    
                            forward declarations
                            
   ######################################################################### */        

// The magnificent kernel!
__global__ void d_simple_particle_swirl( float4* pos, float4* vels, unsigned int N, float dt,
        float3 player_pos); 

// Get our framerate
static double get_framerate();
// Return curr time in ms since last call to this func (high res)
static double get_elapsed();


/* #########################################################################
    
                                initCuda
                                            
        -Sets up both a CUDA context
        -Initializes the shared vertex buffer that we'll use, and 
            gets it registered with CUDA
        
            Return -1 if fail, 0 if success.
   ######################################################################### */
int initCuda(GLuint * vbo) {

    srand(time(0));
    //Set up timer
    LARGE_INTEGER li;
    if(!QueryPerformanceFrequency(&li))
        printf("QueryPerformanceFrequency failed!\n");
    perfFreq = (unsigned long)(li.QuadPart);
    
    //Start off by resetting cudaDevice
    hipDeviceReset();
        
    // first, find a CUDA device and set it to graphic interop
    hipDeviceProp_t  prop;
    int dev;
    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    cudaGLSetGLDevice( dev );
    
    //And set up shared vertex buffer
    glGenBuffers( 1, vbo );
    glBindBuffer( GL_ARRAY_BUFFER, *vbo );
    float4* temppos = (float4*)malloc(BUFFER_SIZE);
    if (!temppos){ printf("Memory alloc error.\n"); return -1;}
    for(int i = 0; i < NUM_PARTICLES; i++)
    {
        /* Initial position in 30-radius ring at y=30 */
        float radius = ((float)rand())/RAND_MAX*27.0+3.0;
        float theta = ((float)rand())/RAND_MAX*2.0*M_PI/8.0;
        temppos[i].x = radius*cosf(theta);
        temppos[i].y = ((float)rand())/RAND_MAX * 1.0 + 29.5;
        temppos[i].z = radius*sinf(theta);
        unsigned char * tmp = (unsigned char *)&(temppos[i].w);
        tmp[0] = (unsigned char) 200;
        tmp[1] = (unsigned char) 200;
        tmp[2] = (unsigned char) 200;
        tmp[3] = 255;
    }
    glBufferData( GL_ARRAY_BUFFER, BUFFER_SIZE, temppos, GL_DYNAMIC_DRAW );
    // (whenever I bind buffer index 0, that's just the way of unbinding
    //     openGL from any buffer...)
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    if (glGetError()){
        unsigned char * glErrorBuffer = (unsigned char *) gluErrorString(glGetError());
        printf("Opengl error: %s\n", glErrorBuffer);
    }
    // register, map and unmap to try to cycle it in...
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(resources, *vbo, cudaGraphicsMapFlagsWriteDiscard) );

    // allocate velocity buffer on device side
    CUDA_SAFE_CALL( hipMalloc( (void**)&d_velocities, BUFFER_SIZE ) );
    float4* tempvel = (float4*)malloc(BUFFER_SIZE);
    if (!tempvel){ printf("Memory alloc error.\n"); return -1;}
    for(int i = 0; i < NUM_PARTICLES; i++)
    {
        /* Initial velocity around origin at 0, 30, 0 */
        tempvel[i].x = ((float)rand())/RAND_MAX * 2.0 - 1.0 + temppos[i].z;
        tempvel[i].y = ((float)rand())/RAND_MAX * 1.0 - 0.5;
        tempvel[i].z = ((float)rand())/RAND_MAX * 2.0 - 1.0 - temppos[i].x;
        tempvel[i].w = 1.0;
    }
    CUDA_SAFE_CALL( hipMemcpy( d_velocities, tempvel, BUFFER_SIZE, hipMemcpyHostToDevice ) );
    free(tempvel);
    free(temppos);

    return 0;
}

/* #########################################################################
    
                             advance_particle_swirl
        - Grab VBO control for CUDA, run kernel to advance particles,
            and pass VBO control back when done.

   ######################################################################### */    
int framesRendered = 0;
void advance_particle_swirl(GLuint * vbo, float px, float py, float pz){
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    CUDA_SAFE_CALL( hipGraphicsMapResources(1, resources) );
    size_t size;
    CUDA_SAFE_CALL( hipGraphicsResourceGetMappedPointer((void **)(&dptr), &size, resources[0]) );
    float dt = (float) get_elapsed();

    // execute the kernel
    if ((framesRendered) > 0)
        d_simple_particle_swirl<<< GRID_SIZE, BLOCK_SIZE >>>(dptr, d_velocities, NUM_PARTICLES, dt,
            make_float3(px, py, pz));

    // unmap buffer object
    CUDA_SAFE_CALL(hipGraphicsUnmapResources(1, resources, 0));
    framesRendered++;
}


/* #########################################################################
    
                           d_simple_particle_swirl
                                    KERNEL!
        
   ######################################################################### */ 
__global__ void d_simple_particle_swirl(float4* pos, float4* vels, unsigned int N, float dt,
        float3 player_pos)
{
    // Indices into the VBO data.
    unsigned long int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N) {
        /* Update vels to orbit <0, 3, 0> */
        float dist2 = pos[i].x*pos[i].x + pos[i].y*pos[i].y + pos[i].z*pos[i].z;
        if (dist2 != 0){
            vels[i].x -= 10.0 * pos[i].x / dist2;
            vels[i].y -= 10.0 * (pos[i].y - 30.0) / dist2;
            vels[i].z -= 10.0 * pos[i].z / dist2;
        }
        /* And update position based on velocity */
        pos[i].x += vels[i].x*dt/1000.0;
        pos[i].y += vels[i].y*dt/1000.0;
        pos[i].z += vels[i].z*dt/1000.0;
        /* Assign color as a rough diffuse lighting model */
        float3 to_our_pos = make_float3(pos[i].x - player_pos.x,
                                                  pos[i].y - player_pos.y,
                                                  pos[i].z - player_pos.z);
        float flen = to_our_pos.x*to_our_pos.x + to_our_pos.y*to_our_pos.y + to_our_pos.z*to_our_pos.z;
        if (flen != 0.0){
            to_our_pos.x /= flen;
            to_our_pos.y /= flen;
            to_our_pos.z /= flen;
        }

        float value = 150;
        unsigned char * tmp = (unsigned char *)&(pos[i]);
        tmp[12] = (unsigned char)(value);
        tmp[13] = (unsigned char)(50); 
        tmp[14] = (unsigned char)(10); 
        tmp[15] = (unsigned char) 150;
    }
}

/* #########################################################################
    
                                get_framerate
                                            
        -Takes totalFrames / ((curr time - start time)/CLOCKS_PER_SEC)
            INDEPENDENT OF GET_ELAPSED; USING THESE FUNCS FOR DIFFERENT
                TIMING PURPOSES
   ######################################################################### */     
static double get_framerate ( ) {
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    unsigned long currTicks = (unsigned long)(li.QuadPart);
    unsigned long elapsed = currTicks - lastTicks_framerate;
    double ret;
    if (elapsed != 0){
        ret = (((double)totalFrames) / (((double)elapsed)/((double)perfFreq)));
    }else{
        ret =  -1;
    }
    totalFrames = 0;
    lastTicks_framerate = currTicks;
    return ret;
}


/* #########################################################################
    
                                get_elapsed
                                            
        -Returns number of milliseconds since last call to this func
            as a double
        
   ######################################################################### */ 
static double get_elapsed(){
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    unsigned long currTicks = (unsigned long)(li.QuadPart);
    double elapsed = (((double)(currTicks - lastTicks_elapsed))/((double)perfFreq))*1000;
    lastTicks_elapsed = currTicks;
    return elapsed;
}
